#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "nn.h"

// ------------------------------------------------------------------------

float* g_weights = NULL;    // store all network weights in one big array.

// ------------------------------------------------------------------------

ConvLayer g_convLayers[16] = {
    { 224,  64,   3,        0,     1728 },
    { 224,  64,  64,     1792,    38656 },    // 2x2 maxpool (224 x 224 -> 112 x 112)
    { 112, 128,  64,    38720,   112448 },
    { 112, 128, 128,   112576,   260032 },    // 2x2 maxpool (112 x 112 -> 56 x 56)
    {  56, 256, 128,   260160,   555072 },
    {  56, 256, 256,   555328,  1145152 },
    {  56, 256, 256,  1145408,  1735232 },
    {  56, 256, 256,  1735488,  2325312 },    // 2x2 maxpool (56 x 56 -> 28 x 28)
    {  28, 512, 256,  2325568,  3505216 },
    {  28, 512, 512,  3505728,  5865024 },
    {  28, 512, 512,  5865536,  8224832 },
    {  28, 512, 512,  8225344, 10584640 },    // 2x2 maxpool (28 x 28 -> 14 x 14)
    {  14, 512, 512, 10585152, 12944448 },
    {  14, 512, 512, 12944960, 15304256 },
    {  14, 512, 512, 15304768, 17664064 },
    {  14, 512, 512, 17664576, 20023872 },    // 2x2 maxpool (14 x 14 -> 7 x 7) -> interpret as flat array
};

DenseLayer g_denseLayers[3] = {
    { 4096, 25088,  20024384, 122784832, false },
    { 4096,  4096, 122788928, 139566144, false },
    { 1000,  4096, 139570240, 143666240, true  },
};

// ------------------------------------------------------------------------

static void evalConv(int idx, const float* bufIn, float* bufOut)
{
    const ConvLayer& layer = g_convLayers[idx];
    const float* W = g_weights + layer.ofsW;
    const float* B = g_weights + layer.ofsB;

    printf("conv %-2d (%3d, %3d, %3d) -> (%3d, %3d, %3d)\n", idx, layer.nIn, layer.sz, layer.sz, layer.nOut, layer.sz, layer.sz);
    fflush(stdout);

    int sz = layer.sz;
    for (int i = 0; i < layer.nOut; i++)
    for (int y = 0; y < sz; y++)
    for (int x = 0; x < sz; x++)
    {
        float sum = B[i];
        for (int j = 0; j < layer.nIn; j++)
        for (int dy = 0; dy < 3; dy++)
        for (int dx = 0; dx < 3; dx++)
        {
            int yy = y + dy - 1;
            int xx = x + dx - 1;
            if (yy >= 0 && yy < sz && xx >= 0 && xx < sz)
                sum += bufIn[sz*sz*j + sz*yy + xx] * W[layer.nIn*3*3*i + 3*3*j + 3*(2-dy) + (2-dx)];
        }
        bufOut[sz*sz*i + sz*y + x] = (sum > 0.f) ? sum : 0.f; // ReLu activation.
    }
}

// ------------------------------------------------------------------------

static void evalDense(int idx, const float* bufIn, float* bufOut)
{
    const DenseLayer& layer = g_denseLayers[idx];
    const float* W = g_weights + layer.ofsW;
    const float* B = g_weights + layer.ofsB;
    float total = 0.f;

    printf("dense %d (%3d) -> (%3d)\n", idx, layer.nIn, layer.nOut);
    fflush(stdout);

    for (int i = 0; i < layer.nOut; i++)
    {
        float sum = B[i];
        for (int j = 0; j < layer.nIn; j++)
            sum += bufIn[j] * W[layer.nIn*i + j];

        if (layer.softmax)
            total += (bufOut[i] = expf(sum));
        else
            bufOut[i] = (sum > 0.f) ? sum : 0.f;
    }

    if (layer.softmax)
        for (int i = 0; i < layer.nOut; i++)
            bufOut[i] *= 1.f / total;
}

// ------------------------------------------------------------------------

#define MAX(a, b) ((a) > (b) ? (a) : (b))
static void maxPool2x2(int sz, int n, const float* bufIn, float* bufOut)
{
    printf("maxpool (%3d, %3d, %3d) -> (%3d, %3d, %3d)\n", n, sz, sz, n, sz/2, sz/2);
    fflush(stdout);

    int h = sz >> 1;
    for (int i = 0; i < n; i++)
    for (int y = 0; y < h; y++)
    for (int x = 0; x < h; x++)
    {
        float v0 = bufIn[sz*sz*i + sz*(y*2)   + (x*2)];
        float v1 = bufIn[sz*sz*i + sz*(y*2)   + (x*2+1)];
        float v2 = bufIn[sz*sz*i + sz*(y*2+1) + (x*2)];
        float v3 = bufIn[sz*sz*i + sz*(y*2+1) + (x*2+1)];
        bufOut[i*h*h + x + h*y] = MAX(MAX(MAX(v0, v1), v2), v3);
    }
}

// ------------------------------------------------------------------------

void evalNetwork(float *buf0) {
    float* buf1 = new float[64 * 224 * 224];

    // Evaluate the network, ping-pong data between buffers.
    printf("Starting inference.\n");
    fflush(stdout);

    evalConv(0, buf0, buf1);
    evalConv(1, buf1, buf0);
    maxPool2x2(224, 64, buf0, buf1);
    evalConv(2, buf1, buf0);
    evalConv(3, buf0, buf1);
    maxPool2x2(112, 128, buf1, buf0);
    evalConv(4, buf0, buf1);
    evalConv(5, buf1, buf0);
    evalConv(6, buf0, buf1);
    evalConv(7, buf1, buf0);
    maxPool2x2(56, 256, buf0, buf1);
    evalConv(8, buf1, buf0);
    evalConv(9, buf0, buf1);
    evalConv(10, buf1, buf0);
    evalConv(11, buf0, buf1);
    maxPool2x2(28, 512, buf1, buf0);
    evalConv(12, buf0, buf1);
    evalConv(13, buf1, buf0);
    evalConv(14, buf0, buf1);
    evalConv(15, buf1, buf0);
    maxPool2x2(14, 512, buf0, buf1);
    evalDense(0, buf1, buf0);
    evalDense(1, buf0, buf1);
    evalDense(2, buf1, buf0);

    printf("Done.\n\n");
    fflush(stdout);

    delete[] buf1;
}

// ------------------------------------------------------------------------

